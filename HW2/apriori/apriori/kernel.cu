#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include <pplinterface.h>
#include "FileReader2.h"
#include "UserTableReader.h"
#include <chrono>
#include <sstream>
using namespace std;

__global__ void centerMatrix(float*userArray, unsigned short *userArrayColumns, unsigned short *userArrayRows)
{
	short column = blockIdx.x * blockDim.x + threadIdx.x + 1; //0th column is empty on purpose
	float cur;
	if (column == 3)
		printf("");
	if (column < *userArrayColumns)
	{
		double total = 0;
		unsigned short count = 0;
		for (short i = 1; i < *userArrayRows; i++)
		{
			cur = userArray[i * (*userArrayColumns) + column];
			if (cur > 0 && cur <= 5)
			{
				total += cur;
				count++;
			}
		}
		float const average = total / count;
		userArray[column] = average;

		for (short i = 1; i < *userArrayRows; i++)
		{
			cur = userArray[i * (*userArrayColumns) + column];
			if (cur > 0 && cur <= 5)
			{
				userArray[i* (*userArrayColumns) + column] = cur - average;
			}
		}
	}
}


__global__ void loadTop5(float*userArray, unsigned short *userArrayRows, unsigned short *userArrayColumns, unsigned short *top5UserArray, unsigned short *top5UserArrayColumns, bool *didSelect)
{
	short const row = blockIdx.x * blockDim.x + threadIdx.x + 1;
	float biggest;
	short biggestIndex = 0;
	if (row < *userArrayRows)
	{
		for (short i = 0; i < *top5UserArrayColumns; i++)
		{
			for (short j = 1; j < *userArrayColumns; j++)
			{
				if (!didSelect[row * (*userArrayColumns) + j])
				{
					if (userArray[row * (*userArrayColumns) + j ] > biggest)
					{
						biggest = userArray[row * (*userArrayColumns) + j];
						biggestIndex = j;
					}
				}
			}
			if (biggestIndex == 0)
				return;
			top5UserArray[row * (*top5UserArrayColumns) + i] = biggestIndex;
			didSelect[row * (*userArrayColumns) + biggestIndex] = true;
			biggestIndex = 0;
			biggest = 0;
		}
	}
}

__global__ void computeSimularMoviesType2TEST(float*userArray, unsigned short *userArrayRows, float*movieArray, unsigned short *movieArrayColumns)
{
	short const movie1 = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short const movie2 = blockDim.y * blockIdx.y + threadIdx.y + 1;
	if (movie1 < *movieArrayColumns && movie2 < *movieArrayColumns && movie1 > movie2)
	{
		for (short i = 1; i < *userArrayRows; i++)	//for every user
		{
			if (userArray[i* (*movieArrayColumns) + movie1] > 1)
				printf("(%d,%f)", i, userArray[i* (*movieArrayColumns) + movie1]);
		}
	}
}

__global__ void computeSimularMoviesType2(float*userArray, unsigned short *userArrayRows, float*movieArray, unsigned short *movieArrayColumns)
{
	short const movie1 = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short const movie2 = blockDim.y * blockIdx.y + threadIdx.y + 1;

	if (movie1 < *movieArrayColumns && movie2 < *movieArrayColumns && movie1 > movie2)
	{
		//printf("%d,%d\n", movie1, movie2);
		double top = 0;
		float topLeft;
		float topRight;
		double bottomLeft = 0;
		double bottomRight = 0;
		for (short i = 1; i < (*userArrayRows); i++)	//for every user
		{
			topLeft = userArray[i* (*movieArrayColumns) + movie1];			//get user rating for movie 1

			topRight = userArray[i* (*movieArrayColumns) + movie2]; 		//get user rating for movie 2					

			top += topRight * topLeft;										//compute this one and add to sum

			bottomLeft += topLeft * topLeft;								//A^2 and add to A's sum
			bottomRight += topRight * topRight;								//B^2 and add to B's sum				
		}

		if (bottomLeft == 0 || bottomRight == 0)
		{
			movieArray[movie1* (*movieArrayColumns) + movie2] = 0;
			movieArray[movie2* (*movieArrayColumns) + movie1] = 0;
		}
		else
		{
			float temp = top / (sqrt(bottomLeft) * sqrt(bottomRight));
			movieArray[movie1* (*movieArrayColumns) + movie2] = temp;
			movieArray[movie2* (*movieArrayColumns) + movie1] = temp;
		}
	}
	if (movie1 < 10 && movie2 < 10 && movie1 >= movie2)
	{
		printf("(movie1, movie2, val)->(%d,%d,%f)\n", movie1, movie2, movieArray[movie1* (*movieArrayColumns) + movie2]);
		printf("(movie2, movie1, val)->(%d,%d,%f)\n", movie2, movie1, movieArray[movie2* (*movieArrayColumns) + movie1]);
	}
	if (movie2 == 9124 && movie1 > 9100)
		printf("movie1,movie2:%d,%d\n", movie1, movie2);
}


//void quicksort(float)

__global__ void computeRecommendedMovies(float*userArray, unsigned short *userArrayColumns, unsigned short *userArrayRows, float*movieArray, bool *didSelect)
{
	short movie = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short user = blockDim.y * blockIdx.y + threadIdx.y + 1;
	float tempSim;
	short selected = 0;
	float top5[6];
	short top5Index[6];
	if (movie < *userArrayColumns && user < *userArrayRows && !didSelect[user* (*userArrayColumns) + movie])
	{
		for (int i = 1; i < *userArrayColumns; i++)
		{
			if (i != movie && didSelect[user * (*userArrayColumns) + i])
			{
				tempSim = movieArray[movie * (*userArrayColumns) + i];
				if (selected < 5)
				{
					top5[5-selected] = tempSim;
					top5Index[5-selected] = i;
					selected++;
				}
				else
				{
					top5[0] = tempSim;
					top5Index[0] = i;
					float temp;
					short temp2;

					//bubble sort......
					for (int i2 = 0; i2 <= 5; i2++)
					{
						for (int j = 0; j < 5; j++)
						{
							if (top5[j] > top5[j + 1] || (top5[j] == top5[j + 1] && top5Index[j] > top5Index[j + 1]))
							{
								temp = top5[j];
								temp2 = top5Index[j];

								top5[j] = top5[j + 1];
								top5Index[j] = top5Index[j + 1];

								top5[j + 1] = temp;
								top5Index[j + 1] = temp2;
							}
						}
					}
				}
			}
		}
		double sum = 0;
		for (int i = 1; i <=selected; i++)
			sum+= top5[i] * movieArray[movie * (*userArrayColumns) + top5Index[i]];
		userArray[user * (*userArrayColumns) + movie] = sum / selected;
		if (movie == 8500 && user > 660)
			printf("user,movie:%d,%d\n", user, movie);
	}
}


void outputData(unsigned short * recomendedMoviesMatrix, unsigned short rows, unsigned short columns, MovieReader m)
{
	fstream f;
	f.open("output.csv", std::fstream::out);
	for (int i = 1; i < rows; i++)
	{
		stringstream ss;
		ss << i;
		for (int j = 0; j < columns; j++)
		{
			ss << ',' << m.movieMapper[recomendedMoviesMatrix[i * columns + j]].movieID;
		}
		f << "user_" << ss.str()<< endl;
	}
	f.close();

}


void populateUserReviewMatrix(float *userReviewMatrix, bool *originalReviewMatrix, UserTableReader r, MovieReader m)
{
	auto vec = r.users;
	for (auto it = vec.begin(); it != vec.end(); ++it)
	{
		for (auto sit = (*it).ratedMovies.begin(); sit != (*it).ratedMovies.end(); ++sit)
		{
			userReviewMatrix[(*it).userID * (m.movieCount +1) + m.movieIDMapper[(*sit).movieID]] = (*sit).rating;
			originalReviewMatrix[(*it).userID * (m.movieCount + 1) + m.movieIDMapper[(*sit).movieID]] = true;
		}
	}
}

hipError_t doAlgo()
{
	printf("----------------------StartedCode-----------------------\n");
	auto t1 = std::chrono::high_resolution_clock::now();
	MovieReader m = MovieReader("movie.csv");
	UserTableReader r = UserTableReader("ratings.csv");

	auto t2 = std::chrono::high_resolution_clock::now();
	printf("-------Filing Reading completed in %d milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count());
	printf("--------------StartedMatrixBuilding---------------------\n");
	auto t3 = std::chrono::high_resolution_clock::now();


	float * movieMatrix = (float *)calloc((m.movieCount + 1) * (m.movieCount + 1), sizeof(float));

	for (int i = 1; i < (m.movieCount + 1); i++)
	{
		movieMatrix[i*(m.movieCount + 1) + i] = 1;
	}
	float * userReviewMatrix = (float *)calloc((r.users.size()+1) * (m.movieCount + 1), sizeof(float));
	bool * originalReviewMatrix = (bool *)calloc( (r.users.size() + 1) * (m.movieCount + 1), sizeof(bool));
	unsigned short * recomendedMoviesMatrix = (unsigned short*)calloc((r.users.size() + 1) * 5, sizeof(unsigned short));
	populateUserReviewMatrix(userReviewMatrix, originalReviewMatrix, r, m);


	auto t4 = std::chrono::high_resolution_clock::now();
	printf("-------matrix created completed in %d milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t4 - t3).count());
	printf("-----------------Started Cuda data copy-----------------\n");
	auto t5 = std::chrono::high_resolution_clock::now();

	string str2;
	hipError_t cudaStatus;
	int movieMatrixColumns = m.movieCount + 1;
	int userReviewColumns = m.movieCount + 1;
	int userReviewRows = r.users.size() + 1;
	unsigned short recomendedMoviesMatrixColumns = 5;
	unsigned short recomendedMoviesMatrixRows = (r.users.size() + 1);

	float * d_movieMatrix;
	cudaStatus = hipMalloc((void**)&d_movieMatrix, sizeof(float) * movieMatrixColumns * movieMatrixColumns);
	cudaStatus = hipMemcpy(d_movieMatrix, movieMatrix, sizeof(float) * movieMatrixColumns * movieMatrixColumns, hipMemcpyHostToDevice);

	unsigned short * d_recomendedMoviesMatrix;
	hipMalloc((void**)&d_recomendedMoviesMatrix, sizeof(unsigned short) * recomendedMoviesMatrixRows * recomendedMoviesMatrixColumns);
	cudaStatus = hipMemcpy(d_recomendedMoviesMatrix, recomendedMoviesMatrix, sizeof(unsigned short)* recomendedMoviesMatrixRows * recomendedMoviesMatrixColumns, hipMemcpyHostToDevice);

	unsigned short * d_recMoviesColumns;
	cudaStatus = hipMalloc((void**)&d_recMoviesColumns, sizeof(unsigned short));
	cudaStatus = hipMemcpy(d_recMoviesColumns, &recomendedMoviesMatrixColumns, sizeof(unsigned short), hipMemcpyHostToDevice);

	unsigned short * d_userReviewMatrixColumns;
	cudaStatus = hipMalloc((void**)&d_userReviewMatrixColumns, sizeof(unsigned short) * 1);
	cudaStatus = hipMemcpy(d_userReviewMatrixColumns,&userReviewColumns,sizeof(unsigned short), hipMemcpyHostToDevice);

	unsigned short * d_userReviewMatrixRows;
	cudaStatus = hipMalloc((void**)&d_userReviewMatrixRows, sizeof(unsigned short) * 1);
	cudaStatus = hipMemcpy(d_userReviewMatrixRows, &userReviewRows, sizeof(unsigned short), hipMemcpyHostToDevice);

	float * d_userReviewMatrix;
	cudaStatus = hipMalloc((void**)&d_userReviewMatrix, sizeof(float) * userReviewRows * userReviewColumns);
	cudaStatus = hipMemcpy(d_userReviewMatrix, userReviewMatrix, sizeof(float)* userReviewRows * userReviewColumns, hipMemcpyHostToDevice);

	
	bool * d_didReviewMatrix;
	cudaStatus = hipMalloc((void**)&d_didReviewMatrix, sizeof(bool) * userReviewRows * userReviewColumns);
	cudaStatus = hipMemcpy(d_didReviewMatrix, originalReviewMatrix, sizeof(bool) * userReviewRows * userReviewColumns, hipMemcpyHostToDevice);

	cudaStatus = hipDeviceSynchronize();

	auto t11 = std::chrono::high_resolution_clock::now();
	auto t12 = std::chrono::high_resolution_clock::now();
	while (std::chrono::duration_cast<std::chrono::milliseconds>(t12 - t11).count() < 20000 )
		t12 = std::chrono::high_resolution_clock::now();


	int blockX = ceil(userReviewRows / 256.0);
	int blockY = ceil(userReviewRows / 16.0);
	int blockXType2 = ceil(userReviewColumns / 256);

	auto t6 = std::chrono::high_resolution_clock::now();
	printf("-------cuda data copy completed in %lld milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t6 - t5).count());
	printf("--------Started Compute Averages for movies-------------\n");
	auto t7 = std::chrono::high_resolution_clock::now();


	centerMatrix<< <blockXType2, 256 >> > (d_userReviewMatrix, d_userReviewMatrixColumns, d_userReviewMatrixRows);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %s after launching computeAverageType2!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	auto t8 = std::chrono::high_resolution_clock::now();
	printf("-------Compute Averages for movies completed in %lld milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t7).count());
	printf("--------Started compute simularMovies-------------\n");
	auto t9 = std::chrono::high_resolution_clock::now();
	float blockDim = 32.0;
	blockX = ceil(movieMatrixColumns / blockDim );
	blockY = ceil(movieMatrixColumns / blockDim);

	computeSimularMoviesType2<<<dim3(blockX, blockY), dim3(blockDim, blockDim) >>>(d_userReviewMatrix, d_userReviewMatrixRows, d_movieMatrix, d_userReviewMatrixColumns);

	cudaStatus = hipGetLastError();
	if (hipSuccess != hipGetLastError())
		printf("Error!\n");
	cudaStatus = hipGetLastError();
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	str2 = hipGetErrorString(cudaStatus);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %s after launching computeSimularMoviesType2!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	blockX = ceil(movieMatrixColumns / 16.0);
	blockY = ceil(userReviewRows / 16.0);

	t8 = std::chrono::high_resolution_clock::now();
	printf("Compute simular movies completed in %lld milliseconds\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t9).count());
	printf("------Started compute recommended movies-----------\n");
	t9 = std::chrono::high_resolution_clock::now();


	computeRecommendedMovies<<<dim3(blockX, blockY), dim3(16, 16) >>>(d_userReviewMatrix, d_userReviewMatrixColumns, d_userReviewMatrixRows, d_movieMatrix, d_didReviewMatrix);
	cudaStatus = hipGetLastError();
	if (hipSuccess != hipGetLastError())
		printf("Error!\n");
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeRecommendedMovies!\n", cudaStatus);
		goto Error;
	}

	t8 = std::chrono::high_resolution_clock::now();
	printf("Compute recommended movies completed in %lld milliseconds\n", std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t9).count());
	blockX = ceil(userReviewRows / 16.0);
	hipError_t cuda3 = hipGetLastError();
	str2 = hipGetErrorString(cuda3);

	loadTop5 << <blockX, 16 >> > (d_userReviewMatrix, d_userReviewMatrixRows, d_userReviewMatrixColumns, d_recomendedMoviesMatrix, d_recMoviesColumns, d_didReviewMatrix);
	hipError_t cuda2 = hipGetLastError();
	str2 = hipGetErrorString(cuda2);
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(recomendedMoviesMatrix, d_recomendedMoviesMatrix, sizeof(unsigned short)* recomendedMoviesMatrixRows * recomendedMoviesMatrixColumns, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Cuda MemCpy failed!!\n", cudaStatus);
		goto Error;
	}
	outputData(recomendedMoviesMatrix, userReviewRows, 5, m);

Error:
	hipFree(d_recMoviesColumns);
	hipFree(d_recomendedMoviesMatrix);
	hipFree(d_didReviewMatrix);
	hipFree(d_movieMatrix);
	hipFree(d_userReviewMatrix);
	hipFree(d_userReviewMatrixColumns);
	hipFree(d_userReviewMatrixRows);

		return cudaStatus;
}

int main()
{

	CUDABackground cuda = CUDABackground();
	doAlgo();
}